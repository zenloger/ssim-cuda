#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>

// Константы для SSIM
#define C1 (0.01f * 0.01f)
#define C2 (0.03f * 0.03f)

// Структура для хранения результатов
struct MatchResult {
    int x;
    int y;
    float score;
};

MatchResult findPatchOnMapCPU(const float* map, const float* patch, 
                            int map_width, int map_height,
                            int patch_width, int patch_height,
                            int step_x = 1, int step_y = 1) {
    // Размер локального окна для SSIM (обычно 8x8 или 11x11)
    const int window_size = 8;
    
    // Размеры корреляционной матрицы
    int grid_width = (map_width - patch_width) / step_x + 1;
    int grid_height = (map_height - patch_height) / step_y + 1;
    
    // Выделение памяти для корреляционной матрицы
    float* correlation = (float*)malloc(grid_width * grid_height * sizeof(float));
    
    // Перебор всех возможных положений патча на карте
    for (int gy = 0; gy < grid_height; ++gy) {
        for (int gx = 0; gx < grid_width; ++gx) {
            // Координаты верхнего левого угла окна на карте
            int map_x = gx * step_x;
            int map_y = gy * step_y;
            
            // Проверка выхода за границы карты
            if (map_x + patch_width > map_width || map_y + patch_height > map_height) {
                correlation[gy * grid_width + gx] = 0.0f;
                continue;
            }
            
            float total_ssim = 0.0f;
            int window_count = 0;
            
            // Перебор всех локальных окон внутри патча
            for (int wy = 0; wy <= patch_height - window_size; wy += 1) {
                for (int wx = 0; wx <= patch_width - window_size; wx += 1) {
                    float sum_map = 0.0f, sum_patch = 0.0f;
                    float sum_map_sq = 0.0f, sum_patch_sq = 0.0f;
                    float sum_map_patch = 0.0f;
                    
                    // Вычисление статистик в пределах локального окна
                    for (int y = 0; y < window_size; ++y) {
                        for (int x = 0; x < window_size; ++x) {
                            float map_val = map[(map_y + wy + y) * map_width + (map_x + wx + x)];
                            float patch_val = patch[(wy + y) * patch_width + (wx + x)];
                            
                            sum_map += map_val;
                            sum_patch += patch_val;
                            sum_map_sq += map_val * map_val;
                            sum_patch_sq += patch_val * patch_val;
                            sum_map_patch += map_val * patch_val;
                        }
                    }
                    
                    // Вычисление SSIM для текущего локального окна
                    float mean_map = sum_map / (window_size * window_size);
                    float mean_patch = sum_patch / (window_size * window_size);
                    float var_map = (sum_map_sq - mean_map * sum_map) / (window_size * window_size);
                    float var_patch = (sum_patch_sq - mean_patch * sum_patch) / (window_size * window_size);
                    float covar = (sum_map_patch - mean_map * sum_patch) / (window_size * window_size);
                    
                    float numerator = (2 * mean_map * mean_patch + C1) * (2 * covar + C2);
                    float denominator = (mean_map*mean_map + mean_patch*mean_patch + C1) * 
                                       (var_map + var_patch + C2);
                    
                    if (denominator != 0) {
                        total_ssim += numerator / denominator;
                        window_count++;
                    }
                }
            }
            
            // Усреднение SSIM по всем локальным окнам
            correlation[gy * grid_width + gx] = window_count > 0 ? total_ssim / window_count : 0.0f;
        }
    }
    
    // Поиск максимального значения SSIM
    MatchResult best_match = {0, 0, 0.0f};
    for (int y = 0; y < grid_height; ++y) {
        for (int x = 0; x < grid_width; ++x) {
            float score = correlation[y * grid_width + x];
            if (score > best_match.score) {
                best_match.x = x * step_x;
                best_match.y = y * step_y;
                best_match.score = score;
            }
        }
    }
    
    free(correlation);
    return best_match;
}

// Функция для конвертации RGB в grayscale
void rgbToGrayscale(const unsigned char* rgb, float* gray, int width, int height, int channels) {
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            int idx = y * width + x;
            int rgb_idx = idx * channels;
            
            float r = rgb[rgb_idx];
            float g = channels > 1 ? rgb[rgb_idx + 1] : r;
            float b = channels > 2 ? rgb[rgb_idx + 2] : r;
            
            gray[idx] = (0.299f * r + 0.587f * g + 0.114f * b) / 255.0f;
        }
    }
}

// Структура для хранения изображения
struct WHDImage {
    uint32_t width;
    uint32_t height;
    uint8_t* data;  // данные в формате uint8_t (0-255) для каждого канала RGB
};

// Функция для чтения WHD файла
WHDImage readWHD(const char* filename) {
    WHDImage img = {0, 0, NULL};
    FILE* file = fopen(filename, "rb");
    
    if (!file) {
        printf("Error: Could not open WHD file %s\n", filename);
        return img;
    }

    // Чтение заголовка (width, height)
    if (fread(&img.width, sizeof(uint32_t), 1, file) != 1 ||
        fread(&img.height, sizeof(uint32_t), 1, file) != 1) {
        printf("Error: Invalid WHD header in %s\n", filename);
        fclose(file);
        return img;
    }

    // Выделение памяти для данных (в формате float для CUDA)
    size_t pixel_count = img.width * img.height * 3;
    img.data = (uint8_t*)malloc(pixel_count * sizeof(uint8_t));
    
    if (!img.data) {
        printf("Error: Memory allocation failed for WHD data\n");
        fclose(file);
        img.width = img.height = 0;
        return img;
    }

    // Чтение пиксельных данных
    uint8_t* pixel_buffer = (uint8_t*)malloc(pixel_count);
    if (!pixel_buffer) {
        printf("Error: Memory allocation failed for pixel buffer\n");
        free(img.data);
        fclose(file);
        img.width = img.height = 0;
        img.data = NULL;
        return img;
    }

    if (fread(pixel_buffer, 1, pixel_count, file) != pixel_count) {
        printf("Error: Invalid pixel data in WHD file %s\n", filename);
        free(pixel_buffer);
        free(img.data);
        fclose(file);
        img.width = img.height = 0;
        img.data = NULL;
        return img;
    }

    for (size_t i = 0; i < pixel_count; i++) {
        img.data[i] = (uint8_t)pixel_buffer[i];
    }
    
    printf("WHD Image: %dx%d\n", img.width, img.height);
    // printf("WHD First 10 pixels: ");
    // for (size_t i = 0; i < 10; i++) {
    //     printf("%d ", img.data[i]);
    // }
    // printf("\n");

    free(pixel_buffer);
    fclose(file);
    return img;
}

// Функция для освобождения памяти WHD изображения
void freeWHD(WHDImage* img) {
    if (img) {
        free(img->data);
        img->width = img->height = 0;
        img->data = NULL;
    }
}

int main(int argc, char** argv) {
    if (argc != 3) {
        printf("Usage: %s <map.whd> <patch.whd>\n", argv[0]);
        return 1;
    }

    // Загрузка изображений
    WHDImage map = readWHD(argv[1]);
    WHDImage patch = readWHD(argv[2]);
    
    if (!map.data || !patch.data) {
        printf("Error loading images\n");
        freeWHD(&map);
        freeWHD(&patch);
        return 1;
    }
    
    // Проверка размеров
    if (patch.width > map.width || patch.height > map.height) {
        printf("Error: Patch must be smaller than map\n");
        freeWHD(&map);
        freeWHD(&patch);
        return 1;
    }
    
    // Конвертация в grayscale
    float* map_gray = (float*)malloc(map.width * map.height * sizeof(float));
    float* patch_gray = (float*)malloc(patch.width * patch.height * sizeof(float));
    
    rgbToGrayscale(map.data, map_gray, map.width, map.height, 3);
    rgbToGrayscale(patch.data, patch_gray, patch.width, patch.height, 3);

    // Поиск участка на карте
    int step = 3; // Шаг скользящего окна (можно изменять)
    MatchResult result = findPatchOnMapCPU(map_gray, patch_gray, 
                                      map.width, map.height,
                                      patch.width, patch.height,
                                      step, step);
    
    // Вывод результатов
    printf("Best match found at (%d, %d) with SSIM score: %f\n", 
           result.x, result.y, result.score);
    printf("Patch dimensions: %dx%d\n", patch.width, patch.height);
    
    // Освобождение памяти
    free(map_gray);
    free(patch_gray);

    // Освобождаем память оригинальных изображений
    freeWHD(&map);
    freeWHD(&patch);
    
    return 0;
}
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>

// Константы для SSIM
#define C1 (0.01f * 0.01f)
#define C2 (0.03f * 0.03f)

// Структура для хранения результатов
struct MatchResult {
    int x;
    int y;
    float score;
};

// Ядро для вычисления SSIM между участком и окном
__global__ void computeSSIMKernel(const float* map, const float* patch, 
                                 float* correlation, int map_width, int map_height,
                                 int patch_width, int patch_height, 
                                 int step_x, int step_y) {
    // Координаты в корреляционной матрице
    int grid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Координаты верхнего левого угла окна на карте
    int map_x = grid_x * step_x;
    int map_y = grid_y * step_y;
    
    // Проверка выхода за границы карты
    if (map_x + patch_width > map_width || map_y + patch_height > map_height) {
        return;
    }

    
    const int window_size = 8; // Размер локального окна для SSIM
    float total_ssim = 0.0f;
    int window_count = 0;
    
    // Проход по всем локальным окнам в пределах участка
    for (int wy = 0; wy <= patch_height - window_size; wy++) {
        for (int wx = 0; wx <= patch_width - window_size; wx++) {
            float sum_map = 0.0f, sum_patch = 0.0f;
            float sum_map_sq = 0.0f, sum_patch_sq = 0.0f;
            float sum_map_patch = 0.0f;
            
            // Вычисление статистик ТОЛЬКО в пределах локального окна
            for (int y = 0; y < window_size; y++) {
                for (int x = 0; x < window_size; x++) {
                    float map_val = map[(map_y + wy + y) * map_width + (map_x + wx + x)];
                    float patch_val = patch[(wy + y) * patch_width + (wx + x)];
                    
                    sum_map += map_val;
                    sum_patch += patch_val;
                    sum_map_sq += map_val * map_val;
                    sum_patch_sq += patch_val * patch_val;
                    sum_map_patch += map_val * patch_val;
                }
            }
            
            // Вычисление SSIM для этого окна
            float mean_map = sum_map / (window_size * window_size);
            float mean_patch = sum_patch / (window_size * window_size);
            float var_map = (sum_map_sq - mean_map * sum_map) / (window_size * window_size);
            float var_patch = (sum_patch_sq - mean_patch * sum_patch) / (window_size * window_size);
            float covar = (sum_map_patch - mean_map * sum_patch) / (window_size * window_size);
            
            float numerator = (2 * mean_map * mean_patch + C1) * (2 * covar + C2);
            float denominator = (mean_map*mean_map + mean_patch*mean_patch + C1) * 
                               (var_map + var_patch + C2);
            
            if (denominator != 0) {
                total_ssim += numerator / denominator;
                window_count++;
            }
        }
    }
    
    int grid_idx = grid_y * ((map_width - patch_width) / step_x + 1) + grid_x;
    correlation[grid_idx] = window_count > 0 ? total_ssim / window_count : 0.0f;
}

// Функция для поиска участка на карте с помощью SSIM
MatchResult findPatchOnMap(const float* h_map, const float* h_patch, 
                          int map_width, int map_height,
                          int patch_width, int patch_height,
                          int step_x = 1, int step_y = 1) {
    // Выделение памяти на устройстве
    float *d_map, *d_patch, *d_correlation;
    size_t map_size = map_width * map_height * sizeof(float);
    size_t patch_size = patch_width * patch_height * sizeof(float);
    
    hipMalloc(&d_map, map_size);
    hipMalloc(&d_patch, patch_size);
    
    // Копирование данных на устройство
    hipMemcpy(d_map, h_map, map_size, hipMemcpyHostToDevice);
    hipMemcpy(d_patch, h_patch, patch_size, hipMemcpyHostToDevice);
    
    // Размеры корреляционной матрицы
    int grid_width = (map_width - patch_width) / step_x + 1;
    int grid_height = (map_height - patch_height) / step_y + 1;
    size_t correlation_size = grid_width * grid_height * sizeof(float);
    
    hipMalloc(&d_correlation, correlation_size);
    
    // Настройка размеров блоков и гридов
    dim3 blockSize(16, 16);
    dim3 gridSize((grid_width + blockSize.x - 1) / blockSize.x, 
                 (grid_height + blockSize.y - 1) / blockSize.y);
    
    // Вычисление корреляционной матрицы
    computeSSIMKernel<<<gridSize, blockSize>>>(d_map, d_patch, d_correlation,
                                             map_width, map_height,
                                             patch_width, patch_height,
                                             step_x, step_y);
    
    // Копирование результата обратно на хост
    float* h_correlation = (float*)malloc(correlation_size);
    hipMemcpy(h_correlation, d_correlation, correlation_size, hipMemcpyDeviceToHost);
    
    // Поиск максимального значения SSIM
    MatchResult best_match = {0, 0, 0.0f};
    for (int y = 0; y < grid_height; ++y) {
        for (int x = 0; x < grid_width; ++x) {
            float score = h_correlation[y * grid_width + x];
            if (score > best_match.score) {
                best_match.x = x * step_x;
                best_match.y = y * step_y;
                best_match.score = score;
            }
        }
    }
    
    // Освобождение памяти
    free(h_correlation);
    hipFree(d_map);
    hipFree(d_patch);
    hipFree(d_correlation);
    
    return best_match;
}

// Функция для конвертации RGB в grayscale
void rgbToGrayscale(const unsigned char* rgb, float* gray, int width, int height, int channels) {
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            int idx = y * width + x;
            int rgb_idx = idx * channels;
            
            float r = rgb[rgb_idx];
            float g = channels > 1 ? rgb[rgb_idx + 1] : r;
            float b = channels > 2 ? rgb[rgb_idx + 2] : r;
            
            gray[idx] = (0.299f * r + 0.587f * g + 0.114f * b) / 255.0f;
        }
    }
}

// Структура для хранения изображения
struct WHDImage {
    uint32_t width;
    uint32_t height;
    uint8_t* data;  // данные в формате uint8_t (0-255) для каждого канала RGB
};

// Функция для чтения WHD файла
WHDImage readWHD(const char* filename) {
    WHDImage img = {0, 0, NULL};
    FILE* file = fopen(filename, "rb");
    
    if (!file) {
        printf("Error: Could not open WHD file %s\n", filename);
        return img;
    }

    // Чтение заголовка (width, height)
    if (fread(&img.width, sizeof(uint32_t), 1, file) != 1 ||
        fread(&img.height, sizeof(uint32_t), 1, file) != 1) {
        printf("Error: Invalid WHD header in %s\n", filename);
        fclose(file);
        return img;
    }

    // Выделение памяти для данных (в формате float для CUDA)
    size_t pixel_count = img.width * img.height * 3;
    img.data = (uint8_t*)malloc(pixel_count * sizeof(uint8_t));
    
    if (!img.data) {
        printf("Error: Memory allocation failed for WHD data\n");
        fclose(file);
        img.width = img.height = 0;
        return img;
    }

    // Чтение пиксельных данных
    uint8_t* pixel_buffer = (uint8_t*)malloc(pixel_count);
    if (!pixel_buffer) {
        printf("Error: Memory allocation failed for pixel buffer\n");
        free(img.data);
        fclose(file);
        img.width = img.height = 0;
        img.data = NULL;
        return img;
    }

    if (fread(pixel_buffer, 1, pixel_count, file) != pixel_count) {
        printf("Error: Invalid pixel data in WHD file %s\n", filename);
        free(pixel_buffer);
        free(img.data);
        fclose(file);
        img.width = img.height = 0;
        img.data = NULL;
        return img;
    }

    for (size_t i = 0; i < pixel_count; i++) {
        img.data[i] = (uint8_t)pixel_buffer[i];
    }
    
    printf("WHD Image: %dx%d\n", img.width, img.height);
    // printf("WHD First 10 pixels: ");
    // for (size_t i = 0; i < 10; i++) {
    //     printf("%d ", img.data[i]);
    // }
    // printf("\n");

    free(pixel_buffer);
    fclose(file);
    return img;
}

// Функция для освобождения памяти WHD изображения
void freeWHD(WHDImage* img) {
    if (img) {
        free(img->data);
        img->width = img->height = 0;
        img->data = NULL;
    }
}

int main(int argc, char** argv) {
    if (argc != 3) {
        printf("Usage: %s <map.whd> <patch.whd>\n", argv[0]);
        return 1;
    }

    // Загрузка изображений
    WHDImage map = readWHD(argv[1]);
    WHDImage patch = readWHD(argv[2]);
    
    if (!map.data || !patch.data) {
        printf("Error loading images\n");
        freeWHD(&map);
        freeWHD(&patch);
        return 1;
    }
    
    // Проверка размеров
    if (patch.width > map.width || patch.height > map.height) {
        printf("Error: Patch must be smaller than map\n");
        freeWHD(&map);
        freeWHD(&patch);
        return 1;
    }
    
    // Конвертация в grayscale
    float* map_gray = (float*)malloc(map.width * map.height * sizeof(float));
    float* patch_gray = (float*)malloc(patch.width * patch.height * sizeof(float));
    
    rgbToGrayscale(map.data, map_gray, map.width, map.height, 3);
    rgbToGrayscale(patch.data, patch_gray, patch.width, patch.height, 3);

    // Поиск участка на карте
    int step = 3; // Шаг скользящего окна (можно изменять)
    MatchResult result = findPatchOnMap(map_gray, patch_gray, 
                                      map.width, map.height,
                                      patch.width, patch.height,
                                      step, step);
    
    // Вывод результатов
    printf("Best match found at (%d, %d) with SSIM score: %f\n", 
           result.x, result.y, result.score);
    printf("Patch dimensions: %dx%d\n", patch.width, patch.height);
    
    // Освобождение памяти
    free(map_gray);
    free(patch_gray);

    // Освобождаем память оригинальных изображений
    freeWHD(&map);
    freeWHD(&patch);
    
    return 0;
}